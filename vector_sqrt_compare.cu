// vector_sqrt_compare.cu
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <chrono>

__global__ void vectorSqrt(const float *A, float *B, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        B[i] = sqrt(A[i]);
    }
}

// CUDAエラーチェックマクロ
#define CUDA_CHECK_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// CPUでの平方根計算
void vectorSqrtCPU(const float *A, float *B, int N) {
    for (int i = 0; i < N; i++) {
        B[i] = sqrt(A[i]);
    }
}

int main() {
    int N = 1<<25; // 33M elements
    size_t size = N * sizeof(float);

    // Allocate memory on the host
    float *h_A = (float*)malloc(size);
    float *h_B_CPU = (float*)malloc(size);
    float *h_B_GPU = (float*)malloc(size);

    // Initialize vectors on the host
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
    }

    // Allocate memory on the device
    float *d_A, *d_B;
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_A, size));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_B, size));

    // Copy vector from host memory to device memory
    CUDA_CHECK_ERROR(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

    // CPUでの計算
    auto start = std::chrono::high_resolution_clock::now();
    vectorSqrtCPU(h_A, h_B_CPU, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_duration = end - start;
    std::cout << "CPU time: " << cpu_duration.count() << " seconds" << std::endl;

    // GPUでの計算
    start = std::chrono::high_resolution_clock::now();
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorSqrt<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    CUDA_CHECK_ERROR(hipGetLastError()); // カーネル呼び出し後のエラーチェック
    CUDA_CHECK_ERROR(hipDeviceSynchronize()); // カーネル呼び出し後の同期
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_duration = end - start;
    std::cout << "GPU time: " << gpu_duration.count() << " seconds" << std::endl;

    // Copy the result from device memory to host memory
    CUDA_CHECK_ERROR(hipMemcpy(h_B_GPU, d_B, size, hipMemcpyDeviceToHost));

    // Verify the result and print some of the computation
    bool verification_passed = true;
    for (int i = 0; i < N; i++) {
        if (i < 10 || i == N-1) { // 最初の10要素と最後の1要素を表示
            std::cout << "Element " << i << ": sqrt(" << h_A[i] << ") = " << h_B_GPU[i] << " (GPU), " << h_B_CPU[i] << " (CPU)" << std::endl;
        }
        if (fabs(h_B_GPU[i] - h_B_CPU[i]) > 1e-5) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            verification_passed = false;
        }
    }

    if (verification_passed) {
        std::cout << "Test PASSED" << std::endl;
    }

    // Free device memory
    CUDA_CHECK_ERROR(hipFree(d_A));
    CUDA_CHECK_ERROR(hipFree(d_B));

    // Free host memory
    free(h_A);
    free(h_B_CPU);
    free(h_B_GPU);

    return 0;
}

